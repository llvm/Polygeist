// clang-format off
// RUN: cgeist %s %stdinclude %cudaopts -O3 -o %s.execm && %s.execm 1 10 2
// RUN: [ "%polymer_pluto_enabled" == "" ] || cgeist %s %stdinclude %polymer_pluto_cudaopts -O3 -o %s.execm && %s.execm 1 10 10 10
// clang-format on
/**
 * seidel-2d.c: This file is part of the PolyBench/C 3.2 test suite.
 *
 *
 * Contact: Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://polybench.sourceforge.net
 */

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

static void init_array(int n, double *A) {
  int i, j;

  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++)
      A[i * n + j] = ((double)i * (j + 2) + 2) / n;
}

static void print_array(int n, double *A)

{
  int i, j;

  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++) {
      fprintf(stderr, "%0.2lf ", A[i * n + j]);
      if ((i * n + j) % 20 == 0)
        fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}

static unsigned num_blocks(int num, int factor) {
  return (num + factor - 1) / factor;
}

__global__ void kernel_stencil(int tsteps, int n, double *A) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
  int j = blockDim.y * blockIdx.y + threadIdx.y + 1;

  if (i < n - 1 && j < n - 1)
    A[i * n + j] =
        (A[(i - 1) * n + j - 1] + A[(i - 1) * n + j] + A[(i - 1) * n + j + 1] +
         A[i * n + j - 1] + A[i * n + j] + A[i * n + j + 1] +
         A[(i + 1) * n + j - 1] + A[(i + 1) * n + j] + A[(i + 1) * n + j + 1]) /
        9;
}

static void kernel(int tsteps, int n, double *A) {
  // FIXME: Parallelizing this should give different results
  const unsigned int threadsPerBlock = 256;

  for (int t = 1; t <= tsteps; t++) {
    dim3 block(threadsPerBlock / 32, 32, 1);
    dim3 grid(num_blocks(n - 2, block.x), num_blocks(n - 2, block.y), 1);
    kernel_stencil<<<grid, block>>>(tsteps, n, A);
  }
}

int main(int argc, char **argv) {

  int dump_code = atoi(argv[1]);
  int n = atoi(argv[2]);
  int tsteps = atoi(argv[3]);

  double *A = (double *)malloc(n * n * sizeof(double));

  init_array(n, A);

  double *dev_A;
  hipMalloc(&dev_A, n * n * sizeof(double));
  hipMemcpy(dev_A, A, n * n * sizeof(double), hipMemcpyHostToDevice);

  kernel(tsteps, n, dev_A);

  hipMemcpy(A, dev_A, n * n * sizeof(double), hipMemcpyDeviceToHost);

  if (dump_code == 1)
    print_array(n, A);

  free((void *)A);
  ;

  return 0;
}
